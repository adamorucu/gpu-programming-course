#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define DataType double
#define NUM_STREAMS 4
#define TPB 32
// #define S_SEG 128

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len, int offset) {
  //@@ Insert code to implement vector addition here
  const int i = blockIdx.x*blockDim.x + threadIdx.x + offset;
  if (i < len)
		out[i] = in1[i] + in2[i];
}

bool identicalArr(DataType *arr1, DataType *arr2, int length) {
    int counter = 0;
    bool identical = true;
    for (int i = 0; i < length; i++) {
        if (arr1[i] != arr2[i]) {
            printf("IND: %d\n", i);
            identical = false;
            counter++;
        }
    }
    return identical;
}

struct timeval t_start, t_end;
void cputimer_start(){
  gettimeofday(&t_start, 0);
}
double cputimer_stop(const char* info){
  gettimeofday(&t_end, 0);
  double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
  printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
  return time;
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;
  hipStream_t streams[NUM_STREAMS];

  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);
  int segSize = atoi(argv[2]);

  // int S_seg = 128;

  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hipHostAlloc(&hostInput1, inputLength * sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc(&hostInput2, inputLength * sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc(&hostOutput, inputLength * sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc(&resultRef, inputLength * sizeof(DataType), hipHostMallocDefault);
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  srand((unsigned)time(NULL));
  for (int i=0; i < inputLength; i++) {
    hostInput1[i] = (DataType)rand() / RAND_MAX * 100;
    hostInput2[i] = (DataType)rand() / RAND_MAX * 100;
    *(resultRef + i) = *(hostInput1 + i) + *(hostInput2 + i);
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength*sizeof(DataType));

  // Start streams
  for (int i = 0; i < NUM_STREAMS; ++i) {
    hipStreamCreate(&streams[i]);
  }

  cputimer_start();
  for (int i = 0; i < inputLength; i+=segSize)
  {
    int len = min(segSize, inputLength - i);
    int gridSize = (len + TPB - 1) / TPB;
    hipStream_t stream = streams[(i/segSize) % NUM_STREAMS];
    
    hipMemcpyAsync(&deviceInput1[i], &hostInput1[i], len * sizeof(DataType), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&deviceInput2[i], &hostInput2[i], len * sizeof(DataType), hipMemcpyHostToDevice, stream);

    vecAdd<<<gridSize, TPB, 0, stream>>>(deviceInput1, deviceInput2, deviceOutput, inputLength, i);

    hipMemcpyAsync(&hostOutput[i], &deviceOutput[i], len * sizeof(DataType), hipMemcpyDeviceToHost, stream);
  }
  cputimer_stop("Calculation");
  
  for (int i = 0; i < NUM_STREAMS; ++i) {
    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);
  }

  //@@ Insert code below to compare the output with the reference
  bool identical = identicalArr(resultRef, hostOutput, inputLength);
  if (identical)
    printf("Identical arrays\n");
  else
    printf("NOT IDENTICAL\n");

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  hipHostFree(hostInput1);
  hipHostFree(hostInput2);
  hipHostFree(hostOutput);
  hipHostFree(resultRef);

  return 0;
}
