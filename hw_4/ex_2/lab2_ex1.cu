#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  // *(out + i) = *(in1 + i) + *(in2 + i);
  out[i] = in1[i] + in2[i];
}

bool identicalArr(DataType *arr1, DataType *arr2, int length) {
    int counter = 0;
    bool identical = true;
    for (int i = 0; i < length; i++) {
        if (arr1[i] != arr2[i]) {
            printf("IND: %d\n", i);
            identical = false;
            counter++;
        }
    }
    return identical;
}

struct timeval t_start, t_end;
void cputimer_start(){
  gettimeofday(&t_start, 0);
}
double cputimer_stop(const char* info){
  gettimeofday(&t_end, 0);
  double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
  printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
  return time;
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);

  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType*)malloc(inputLength*sizeof(DataType));
  hostInput2 = (DataType*)malloc(inputLength*sizeof(DataType));
  hostOutput = (DataType*)malloc(inputLength*sizeof(DataType));
  resultRef = (DataType*)malloc(inputLength*sizeof(DataType));
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  srand((unsigned)time(NULL));
  for (int i=0; i < inputLength; i++) {
    hostInput1[i] = (DataType)rand() / RAND_MAX * 100;
    hostInput2[i] = (DataType)rand() / RAND_MAX * 100;
    *(resultRef + i) = *(hostInput1 + i) + *(hostInput2 + i);
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength*sizeof(DataType));

  // Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  cputimer_start();

  //@@ Insert code to below to Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength*sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength*sizeof(DataType), hipMemcpyHostToDevice);

  //@@ Initialize the 1D grid and block dimensions here
  const int TPB = 32;
  const int BPG = (inputLength + TPB - 1)/TPB;

  //@@ Launch the GPU Kernel here
  vecAdd<<<BPG, TPB>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(DataType), hipMemcpyDeviceToHost);

  cputimer_stop("Calculation");

  //@@ Insert code below to compare the output with the reference
  bool identical = identicalArr(resultRef, hostOutput, inputLength);
  if (identical)
    printf("Identical arrays\n");
  else
    printf("NOT IDENTICAL\n");

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  return 0;
}
